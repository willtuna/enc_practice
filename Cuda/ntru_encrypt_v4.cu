#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>//**************************************************
#include <hip/hip_runtime.h>//**************************************************

#define NUM_TRITS 41

typedef struct trits {
    int trit_poly [NUM_TRITS] ;
} Message;

int char2trit(char * infile_path, Message **msg_arr);
int trit2char(Message *const decrypted_msg_arr, int num_block);
void read_tritfile(FILE *ptr);




//#define DBG 0
#define N  251
int p = 3;
int q =128;
int d = 2;

int f        []=  {-1,0,0,-1,1,0,1,0,0,1,-1,0,0,1,0,-1,1,1,0,1,0,-1,0,0,0,0,-1};
int g        []=  {0,1,0,-1,0,0,1,0,0,-1,1,0,0,0,1,0,0,0,0,0,0,1,1,0,0,0,-1,-1,-1,-1};

int f_inv_q  []= { 43,118,82,111,124,16,107,97,61,34,60,48,52,18,25,92,83,32,34,36,9,81,3,111,41,45,80,70,30,65,124,51,67,35,103,44,101,102,3,19,98,97,120,85,59,94,67,104,92,18,34,76,72,70,119,100,112,0,93,91,18,31,74,72,65,69,122,88,40,66,72,48,68,55,39,58,67,89,68,110,98,107,110,62,92,56,58,34,83,31,39,35,93,125,40,127,105,53,118,12,50,119,114,127,55,19,25,67,61,80,85,70,114,121,109,14,77,36,91,81,48,50,54,61,14,12,86,91,22,54,70,104,95,26,42,75,93,21,80,47,34,89,52,71,81,66,20,17,63,7,61,100,125,25,51,78,97,27,109,50,7,26,21,11,77,100,113,83,27,78,35,34,27,123,25,54,95,77,83,108,38,68,6,26,0,38,50,55,87,33,75,31,24,29,65,75,39,81,50,21,85,88,2,63,126,82,23,110,90,47,12,78,1,93,9,122,19,35,126,1,22,71,86,28,6,13,113,5,125,40,83,53,111,44,76,108,51,55,101,40,1,93,16,87,43,103,18,97,72,65,2 };

int f_inv_p  []= { 2,2,1,2,0,1,2,0,1,1,0,1,2,1,0,2,2,1,1,0,2,0,1,0,0,0,0,2,1,1,2,1,0,0,1,0,2,1,2,0,0,2,1,0,1,0,2,1,2,0,0,0,0,2,2,2,0,2,0,1,0,0,1,0,1,1,0,0,0,1,1,0,2,1,0,1,0,1,0,0,1,2,1,1,0,0,1,1,0,2,0,1,0,2,2,0,0,2,0,0,2,2,0,0,0,2,1,0,0,2,0,2,0,1,2,0,0,1,0,0,1,1,1,0,0,0,0,1,1,0,0,0,0,2,2,2,0,2,0,2,0,2,1,2,0,2,0,1,2,1,2,1,0,1,0,1,1,1,1,0,1,0,2,0,0,0,2,0,1,2,2,1,0,1,2,1,1,2,0,1,0,1,1,1,0,1,1,1,2,1,1,0,1,2,2,2,0,2,1,0,1,0,0,2,1,1,2,2,2,1,0,2,0,1,1,1,1,2,1,1,1,2,0,0,0,1,2,1,0,2,2,1,1,0,1,0,0,0,0,1,0,0,2,2,2,1,2,2,1,2,2 };


int rand_sel []=  {0,0,-1,-1,0,1,0,0,-1,1,1,1,0,1,0,-1,-1,1,1,0,-1,0,1,0,0,0,1,0,1,-1,-1,0,0,-1,0,0,0,0,-1};

int irr_l    []=  { -1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1
};
int key_pub [] = { 0,129,354,117,-21,126,-156,303,489,66,-42,153,276,228,546,102,678,759,429,-123,444,609,1152,759,750,1098,429,294,36,-48,-879,30,-774,-375,-576,-327,33,516,156,-567,300,153,339,-81,141,-243,444,612,714,-87,-198,312,183,207,78,246,15,123,3,-129,3,3,174,-324,426,195,390,27,153,-123,-327,-744,12,18,-462,-231,462,564,150,-120,159,-288,-705,-129,-249,-84,210,321,243,270,33,-132,-90,-180,333,-417,18,36,195,300,87,69,747,591,-96,-510,-390,-360,-75,-24,-459,-447,-222,204,555,441,90,645,396,84,93,-987,-384,126,360,-174,-15,-201,147,-714,-666,-174,447,786,171,-78,-6,240,-639,-762,-366,-300,9,-165,-12,-144,-102,-531,66,306,18,315,525,342,-159,-351,147,267,6,-435,-255,-567,369,-204,-54,294,768,-477,-213,-360,-219,-186,333,300,252,174,-6,345,-279,237,84,-369,-696,45,-696,-300,237,768,345,696,39,207,-363,-522,-264,-69,-66,-78,246,-69,243,267,-9,-546,-561,-375,-546,-492,27,435,357,627,105,-93,171,213,-462,177,120,867,-438,354,156,180,-594,300,-111,123,447,309,-393,-669,318,-618,-240,-576,639,-450,195,123,450,-741,438,150,-126,90,-123,-84,255,633,-168,465,-123,618,-264,240,-390,-90,-393,93,-369,-603,-339,-348,-630,-351,-153,-342,-105,-84,-306,-546,-747,-783,-870,-756,-708,-417,-201,-6 };


typedef struct Poly Poly;// forward declaration

typedef void  (*func_p)(Poly *);
typedef void  (*func_p_arr_i)(Poly *,int [], int);
typedef Poly* (*func_p_i)(Poly*, int);
typedef Poly* (*func_p_p_i)(Poly *, Poly *, int);
typedef Poly* (*func_p_p_p_i)(Poly *, Poly *, Poly *,int);

typedef struct Cipher {
    int Cipher_poly[251] ;//should be initialized
    int degree;
} Cipher;

struct Poly {
    int * coef;
    int degree;
    func_p free,print;
    func_p_i scalar_mult,center_lift;
    func_p_p_i add;
    func_p_p_p_i mult;
    func_p_arr_i set;
};


int table_size = 1000;
int  poly_count = 0;
__host__ __device__ int Poly_init(Poly** self);

__host__ __device__ void Poly_free(Poly* ptr);
__host__ __device__ void Poly_print(Poly* self);
__host__ __device__ Poly * Poly_CenterLift(Poly *ptr_a, int q);
__host__ __device__ Poly * Poly_scalar_mult(Poly * self, int multiplier);
__host__ __device__ Poly * Poly_add(Poly * self, Poly *ptr_b, int field_N);
__host__ __device__ Poly * Poly_mult(Poly *self, Poly *ptr_b, Poly* ptr_irr,int q);
__host__ __device__ void Poly_set(Poly *self, int [], int size); 
__host__ void File_export(FILE * fptr_out, Cipher* self);

__host__ __device__ void Cipher_set(int * arr, Poly *self);

/* ref 
print("encryption ...")
r_ccov_h = p*P(rand_sel)*P(key_pub)+P(message)
cipher  = poly_ring_mult_over_q_with_irr(poly1_l=r_ccov_h.coef,poly2_l=[1],irr_l=irr_l,q=q)
print('cipher:',cipher)
*/

__global__ void encrypt(Message * arr_trit_msg, int num_block, Cipher * poly_cipher_arr, Poly * poly_irr_l, Poly * poly_key_pub, Poly * poly_rand_sel, int p, int q){
	//int blk_idx = blockIdx.x*blockDim.x+threadIdx.x;
	int blk_idx=0;
	int TotalThread = blockDim.x*gridDim.x;
	int stripe = num_block / TotalThread;
	int head   = (blockIdx.x*blockDim.x + threadIdx.x)*stripe;
	int LoopLim = head+stripe;

	Poly* poly_message; 
	Poly_init(& poly_message);
    Poly* poly_scalmul ;
    Poly* poly_mult    ;
    Poly* poly_cipher  ;
	
	for(blk_idx=head ; blk_idx<LoopLim ; blk_idx++ ){
		if(poly_message -> coef != NULL){// has previous message
			poly_message -> free(poly_message); 
			poly_scalmul -> free(poly_scalmul); 
			poly_mult    -> free(poly_mult   ); 
			poly_cipher  -> free(poly_cipher ); 
		}
		poly_message -> set (poly_message  , arr_trit_msg[blk_idx].trit_poly, NUM_TRITS);
                int i=0;
		for(i = 0; i <= NUM_TRITS; i++){
			if(poly_message -> coef[i] < 0){
				poly_message -> coef[i] += 3;
			}
		}
		poly_scalmul = poly_rand_sel -> scalar_mult(poly_rand_sel, p) ;
		poly_mult    = poly_scalmul  -> mult(poly_scalmul, poly_key_pub, poly_irr_l, q);
		poly_cipher  = poly_mult -> add ( poly_mult, poly_message, q);


		Cipher_set(poly_cipher_arr[blk_idx].Cipher_poly, poly_cipher);
		poly_cipher_arr[blk_idx].degree = poly_cipher -> degree;
		//弄成array type => poly_cipher_arr[blk_idx]
		//File_export(fptr_out,poly_cipher)移到main;
	}
}

float GPU_kernel(Message * arr_trit_msg, int num_block, Cipher * poly_cipher, Poly * poly_irr_l, Poly * poly_key_pub, Poly * poly_rand_sel, int p, int q){
	
	Message * darr_trit_msg;
	Poly * dpoly_irr_l, * dpoly_key_pub, * dpoly_rand_sel;
        int * dpoly_irr_l_coef   ,* dpoly_key_pub_coef ,* dpoly_rand_sel_coef;


	Cipher * dpoly_cipher;

	//int * poly_irr_l_coef, 
	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device******************
	hipMalloc((void**)&darr_trit_msg      ,sizeof(Message)*num_block);
	hipMalloc((void**)&dpoly_irr_l        ,sizeof(Poly)*1);
	hipMalloc((void**)&dpoly_irr_l_coef   ,sizeof(int)*(poly_irr_l->degree+1));
	hipMalloc((void**)&dpoly_key_pub      ,sizeof(Poly)*1);
	hipMalloc((void**)&dpoly_key_pub_coef ,sizeof(int)*(poly_key_pub->degree+1));
	hipMalloc((void**)&dpoly_rand_sel     ,sizeof(Poly)*1);
	hipMalloc((void**)&dpoly_rand_sel_coef,sizeof(int)*(poly_rand_sel->degree+1));
	hipMalloc((void**)&dpoly_cipher       ,sizeof(Cipher)*num_block);//initialize
	// Allocate Memory Space on Device (for observation)
	//hipMalloc((void**)&dInd,sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated********************
	hipMemcpy(darr_trit_msg   , arr_trit_msg   , sizeof(Message)*num_block, hipMemcpyHostToDevice);
	hipMemcpy(dpoly_irr_l     , poly_irr_l     , sizeof(Poly)*1           , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_key_pub   , poly_key_pub   , sizeof(Poly)*1           , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_rand_sel  , poly_rand_sel  , sizeof(Poly)*1           , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_irr_l_coef    , poly_irr_l->coef   , sizeof(int)*(poly_irr_l->degree+1)   , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_key_pub_coef  , poly_key_pub->coef , sizeof(int)*(poly_key_pub->degree+1) , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_rand_sel_coef , poly_rand_sel->coef, sizeof(int)*(poly_rand_sel->degree+1), hipMemcpyHostToDevice);
	dpoly_irr_l   ->coef = dpoly_irr_l_coef;
	dpoly_key_pub ->coef = dpoly_key_pub_coef;
	dpoly_rand_sel->coef = dpoly_rand_sel_coef;
	

	// Copy Data (indsave array) to device
	//hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	
	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid (2);
	dim3 dimBlock(4);
	encrypt<<<dimGrid,dimBlock>>>(darr_trit_msg, num_block, dpoly_cipher, dpoly_irr_l, dpoly_key_pub, dpoly_rand_sel,p,q);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back******************
	hipMemcpy(poly_cipher, dpoly_cipher, sizeof(Cipher)*num_block, hipMemcpyDeviceToHost);
	
	// Release Memory Space on Device
	hipFree(darr_trit_msg);
	hipFree(dpoly_irr_l);
	hipFree(dpoly_irr_l_coef);
	hipFree(dpoly_key_pub);
	hipFree(dpoly_key_pub_coef);
	hipFree(dpoly_rand_sel);
	hipFree(dpoly_rand_sel_coef);
	hipFree(dpoly_cipher);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}

int main(int argc , char** argv){

    char * infilepath  =  argv[1];
    char * outfilepath =  argv[2];

    Message * arr_trit_msg = NULL;
    int num_block=0;
    FILE * fptr_out = NULL;
    fptr_out = fopen(outfilepath,"w");
	
    if( (num_block = char2trit(infilepath, &arr_trit_msg)) == -1 ){
        printf("Error\n");
        return 1;
    }
    // param shared
    Poly * poly_irr_l  ; 
    Poly_init(& poly_irr_l   );
    poly_irr_l      -> set (poly_irr_l    ,irr_l    ,sizeof(irr_l    )/sizeof(int));
    // param decryption
    Poly * poly_g;
    Poly * poly_f      ; 
    Poly * poly_f_inv_q; 
    Poly * poly_f_inv_p; 

    Poly_init(& poly_g       );
    Poly_init(& poly_f       );
    Poly_init(& poly_f_inv_q );
    Poly_init(& poly_f_inv_p );

    poly_g          -> set (poly_g        ,g        ,sizeof(g        )/sizeof(int));
    poly_f          -> set (poly_f        ,f        ,sizeof(f        )/sizeof(int));
    poly_f_inv_q    -> set (poly_f_inv_q  ,f_inv_q  ,sizeof(f_inv_q  )/sizeof(int));
    poly_f_inv_p    -> set (poly_f_inv_p  ,f_inv_p  ,sizeof(f_inv_p  )/sizeof(int));
    // param eyncryption
    Poly * poly_key_pub; 
    //Poly * poly_message; 
    Poly * poly_rand_sel;

    Poly_init(& poly_key_pub );

    Poly_init(& poly_rand_sel);

    poly_key_pub    -> set (poly_key_pub  ,key_pub  ,sizeof(key_pub  )/sizeof(int));
//    poly_message    -> set (poly_message  ,message  ,sizeof(message  )/sizeof(int));
    poly_rand_sel   -> set (poly_rand_sel ,rand_sel ,sizeof(rand_sel )/sizeof(int));

	/* CPU side*/
	
	Cipher * arr_cipher_cpu = (Cipher *)malloc(sizeof(Cipher)*num_block);
	
	/* GPU side*/
	float elapsedTime = GPU_kernel(arr_trit_msg, num_block, arr_cipher_cpu, poly_irr_l, poly_key_pub, poly_rand_sel, p , q);
	
	printf("GPU time = %5.2f ms\n", elapsedTime);
	
	for( int blk_idx =0; blk_idx < num_block ; ++blk_idx  ){
	File_export(fptr_out,&arr_cipher_cpu[blk_idx]);
	}
	
	free(arr_cipher_cpu);
//  Encryption intermediate ptr

    /*Poly* poly_scalmul ;
    Poly* poly_mult    ;
    Poly* poly_cipher  ;
    for( int blk_idx =0; blk_idx < num_block ; ++blk_idx  ){
        if(poly_message -> coef != NULL){// has previous message
             poly_message -> free(poly_message); 
             poly_scalmul -> free(poly_scalmul); 
             poly_mult    -> free(poly_mult   ); 
             poly_cipher  -> free(poly_cipher ); 
        }
        poly_message -> set (poly_message  , arr_trit_msg[blk_idx].trit_poly, NUM_TRITS);
        poly_scalmul = poly_rand_sel -> scalar_mult(poly_rand_sel, p) ;
        poly_mult    = poly_scalmul  -> mult(poly_scalmul, poly_key_pub, poly_irr_l, q);
        poly_cipher  = poly_mult -> add ( poly_mult, poly_message, q);
        File_export(fptr_out,poly_cipher);
    }*/
//    printf("message:  ");
//    poly_message -> print(poly_message);
//    Poly* poly_scalmul = poly_rand_sel -> scalar_mult(poly_rand_sel, p) ;
//    Poly* poly_mult    = poly_scalmul  -> mult(poly_scalmul, poly_key_pub, poly_irr_l, q);

//    printf("mult:  ");
//    poly_mult -> print(poly_mult);

//    Poly* poly_cipher = poly_mult -> add ( poly_mult, poly_message, q);
//    printf("Cipher:  ");
//    poly_cipher -> print(poly_cipher);

/*
    printf("Decryption:\n");
    Poly* poly_fq_mult_e = poly_cipher -> mult( poly_cipher, poly_f , poly_irr_l, q );
    printf("Poly fq mult cipher\n");
    poly_fq_mult_e -> print(poly_fq_mult_e);
    printf("Center Lifting\n");
    Poly* poly_centerlift_q = poly_fq_mult_e -> center_lift(poly_fq_mult_e, q);
    poly_centerlift_q -> print(poly_centerlift_q);
    printf("Poly f_inv_q mult center_lift q\n");
    Poly* f_inv_p_center =  poly_centerlift_q  -> mult( poly_centerlift_q, poly_f_inv_p, poly_irr_l, p);
    f_inv_p_center -> print(f_inv_p_center);
    printf("Center Lifting\n");
    Poly* decrypted =  f_inv_p_center -> center_lift(f_inv_p_center, p);
    decrypted -> print(decrypted);
*/


    // free the polynomial
    poly_irr_l    -> free(poly_irr_l    ); 
    poly_g        -> free(poly_g        );
    poly_f        -> free(poly_f        ); 
    poly_f_inv_q  -> free(poly_f_inv_q  ); 
    poly_f_inv_p  -> free(poly_f_inv_p  ); 
    poly_key_pub  -> free(poly_key_pub  ); 
    poly_rand_sel -> free(poly_rand_sel );

    fclose(fptr_out);

    return 0;
}





/* ref
typedef struct {
    int * coef;
    int degree;
    func_p free,
           print;
    func_p_i scalar_mult;
    func_p_p_i add;
    func_p_p_p_i mult;
}Poly;
*/


__host__ __device__ int Poly_init(Poly** self){
    if(NULL == (*self= (Poly*)malloc(sizeof(Poly))) ) return EXIT_FAILURE;

    (*self) ->coef   = NULL;
    (*self) ->degree = 0;

    (*self) ->free = Poly_free;   
    (*self) ->print = Poly_print;
    (*self) ->scalar_mult = Poly_scalar_mult;
    (*self) ->add   = Poly_add;
    (*self) ->mult  = Poly_mult;
    (*self) ->set   = Poly_set;
    (*self) ->center_lift= Poly_CenterLift;
    return 0;
}




 __device__ __host__ void Poly_free(Poly* ptr){
    if(ptr->coef != NULL ){
        free(ptr->coef);
        ptr->coef = NULL;
        ptr->degree = 0;
    }
}
 __device__ __host__ void Poly_print(Poly* self){
    //printf("Poly Coef:   ");
    //printf("{");
    for(int idx=0; idx <= self->degree ; ++idx){
        if(idx != self->degree ){
            printf("%d ",self ->coef[idx]);
        }
        else{
            printf("%d\n",self->coef[idx]);
        }
    }
}

__host__ void File_export(FILE * fptr_out, Cipher* self){
    for(int idx=0; idx <= self->degree ; ++idx){
        if(idx != self->degree ){
            fprintf(fptr_out,"%d ",self ->Cipher_poly[idx]);
        }
        else{
            fprintf(fptr_out,"%d\n",self->Cipher_poly[idx]);
        }
    }
}

 __device__ __host__ Poly * Poly_scalar_mult(Poly * self, int multiplier){
    Poly * rtn;
    Poly_init(&rtn);   
    rtn -> coef = (int*)malloc( sizeof(int)*(self->degree+1) );
    rtn -> degree = self->degree;
    for(int idx=0;idx <= self->degree ; ++idx){
        rtn->coef[idx]= (multiplier * self->coef[idx])%N;
    }
    return rtn;
}

 __device__ __host__ Poly * Poly_add(Poly * ptr_a, Poly *ptr_b, int field_N){
    Poly * large_ptr = (ptr_a -> degree > ptr_b -> degree ) ? ptr_a  : ptr_b ;
    Poly * small_ptr = (ptr_a -> degree > ptr_b -> degree ) ? ptr_b : ptr_a;

    Poly * rtn;
    Poly_init(&rtn);   
    rtn -> coef = (int*)malloc( sizeof(int)*(large_ptr->degree+1) );
    rtn -> degree = large_ptr->degree;

    for(int idx =0 ; idx <= large_ptr -> degree ; ++idx){
        if ( idx > small_ptr-> degree)
            rtn -> coef[idx] = large_ptr -> coef[idx];
        else
            rtn -> coef[idx] = (large_ptr->coef[idx]+ small_ptr->coef[idx]) % field_N;
    }
    return rtn;
}

__device__ __host__ Poly * Poly_CenterLift(Poly *ptr_a, int q){

    Poly * rtn;
    Poly_init(&rtn);   
    rtn -> coef = (int*)malloc( sizeof(int)*(ptr_a->degree + 1) );
    rtn -> degree = ptr_a -> degree;

    int tmp;
    for(int i =0 ; i <= (ptr_a -> degree) ; ++i){
        tmp = ptr_a->coef[i];
        if( tmp > (q/2))
            rtn -> coef[i] = tmp - q;
        else 
            rtn -> coef[i] = tmp ;
    }
    return rtn;
}

__device__ __host__ Poly * Poly_mult(Poly *ptr_a, Poly *ptr_b, Poly* ptr_irr,int q){

    int rtn_idx = 0;
    int size = ptr_irr -> degree;
    int tmp;


    Poly * poly_rtn;
    Poly_init(& poly_rtn);
    poly_rtn -> coef = (int*)malloc(sizeof(int)*size);
    poly_rtn -> degree = size-1;

#ifdef DBG
    printf("poly rnt:"); Poly_print(poly_rtn);
    printf("ptr_a");     Poly_print(ptr_a);
    printf("ptr_b");     Poly_print(ptr_b);
    printf("ptr_irr");   Poly_print(ptr_irr);
#endif

    for (int idx = 0 ; idx <= ptr_a -> degree ; ++idx){
        for(int idy = 0 ; idy <= ptr_b -> degree ; ++ idy){
            rtn_idx = (idx + idy)%size;
#ifdef DBG
            printf("\n(%d + %d * %d )mod N ",  poly_rtn -> coef[rtn_idx], ptr_a -> coef[idx], ptr_b->coef[idy]);
#endif
            tmp  = (poly_rtn->coef[rtn_idx] + ptr_a->coef[idx] * ptr_b->coef[idy])%q;
            if(tmp <0)
                poly_rtn->coef[rtn_idx] = tmp + q;
            else
                poly_rtn->coef[rtn_idx] = tmp;
#ifdef DBG
            printf("poly_rtn[%d] = %d \n", rtn_idx, poly_rtn -> coef[rtn_idx]);
#endif
        }
    }
    // update degree
    for (int idx = size-1 ; idx >= 0 ; idx--){
	if(poly_rtn->coef[idx] != 0 ){
	    poly_rtn->degree = idx;
	    break;
        }
    }
    return poly_rtn;
}
__device__ __host__ void Poly_set(Poly *self, int arr[], int size){
    for (int idx=size-1 ; idx >=0 ; -- idx){
        if(arr[idx] != 0){
            self -> degree = idx;
            break;
        }
    }
    self->coef = (int*)malloc(sizeof(int)*size );
    for (int i = 0 ; i <= self->degree ; ++i){
        self->coef[i]=arr[i];
    }
} 

__device__ void Cipher_set(int * arr, Poly *self){
    for (int i = 0 ; i < 251 ; ++i){
		if(i <= self->degree){
			arr[i]=self->coef[i];
		}else {
			arr[i]=0;
		}
    }
	
} 




//              char2trit 
int char2trit(char * infile_path, Message ** msg_arr){
    FILE *infile_p;
    long int filesize = 0;

    infile_p = fopen(infile_path ,"r");
    if(infile_p == NULL){
        printf("Error Opeing Files %s\n", infile_path);
        return -1;
    }
    else{
        long int start_pos,end_pos;
        printf("Opeing Files :%s   Success !\n", infile_path);
        fseek(infile_p,0L,SEEK_END);
        end_pos   = ftell(infile_p);
        fseek(infile_p,0L,SEEK_SET);
        start_pos = ftell(infile_p);
        printf("File End Pos  : %ld\nFile Start Pos: %ld\n",end_pos,start_pos);
        filesize = end_pos -start_pos;
        printf("File Size     : %ld\n",filesize);
    }
    char char_read;
    unsigned long long int tmp_8byte;
    unsigned long long int * block_array8b_ptr;

    // filesize is number of byte
    // Our array element needs chunk it into 8-byte block
    // Number of block = filesize/8+1
    int num_block = (filesize%8 == 0 )? filesize/8 : filesize/8+1;
    printf("Number of Block: %d \n",num_block);
    // calloc to clean the malloc
    block_array8b_ptr = (unsigned long long *)calloc(num_block,sizeof(unsigned long long int));

    int chr_count = 0;
    int blk_idx = 0;
    while( (char_read = fgetc(infile_p))  != EOF){
       printf("%c",char_read);
       tmp_8byte = (chr_count == 0) ? 0 :tmp_8byte << 8;
       tmp_8byte = tmp_8byte | char_read; // shift one char size(8 bit) then bit wise or

       chr_count = (chr_count+1) % 8 ;

       if (chr_count == 0){
           block_array8b_ptr[blk_idx] = tmp_8byte;// store tmp code into array
           blk_idx ++;
       }
    }
    // Padding Last Word with 0
    if (blk_idx == num_block-1){
           while(chr_count != 0){ 
               tmp_8byte = tmp_8byte << 8;
               chr_count = (chr_count+1) % 8 ;
           }
           block_array8b_ptr[blk_idx] = tmp_8byte;// store last word into block array 
        blk_idx++;
    }
    if (blk_idx != num_block){
        printf("Block Segmentation Fault !!! Debug\n");
        return -1;
    }

    // Opeing Memory for trits
    *msg_arr = (Message *)calloc(num_block,sizeof(Message));

    // Turn into trits
    for (int b_idx = 0 ; b_idx < num_block ; ++ b_idx){ // read out block
        tmp_8byte = block_array8b_ptr[b_idx];
        printf("\nblk_idx %d : %llu\n",b_idx,tmp_8byte);

        for(int t_idx =0 ; t_idx < NUM_TRITS ; ++t_idx ){ // encode to trits and write into trits
    // Convert trit {0 1 2} to  0 1 -1
            int trit = (tmp_8byte % 3);
            trit = (trit == 2)? -1 : trit;
            (*msg_arr)[b_idx].trit_poly[t_idx] = trit;
            tmp_8byte /= 3;
        } 
    }

    fclose(infile_p);
    // Finish Encoding to Trits
    return num_block;
}
    


__host__ __device__  int trit2char(Message *const decrypted_msg_arr, int num_block){
    unsigned long long int tmp_8byte_decode;
    printf("Decoding From Trits to char\n");
    for (int b_idx = 0 ; b_idx < num_block ; ++ b_idx){ // read out block
            tmp_8byte_decode = 0;
        for(int t_idx = NUM_TRITS-1 ; t_idx >= 0  ; --t_idx ){ // encode to trits and write into trits
            tmp_8byte_decode *= 3;
            tmp_8byte_decode += decrypted_msg_arr[b_idx].trit_poly[t_idx]; 
        } 

//        printf("b_idx: %d: ", b_idx);
//        printf("tmp_8byte : %llu  :",tmp_8byte_decode);
        char chr_tmp;
        for(int shf_idx =7; shf_idx >= 0 ; --shf_idx){// take out 8 bit by 8 bit
            chr_tmp =  tmp_8byte_decode >> (shf_idx*8) & 0xff; // after shifting , taking 8 bit out
            printf("%c",chr_tmp);
        }
        printf("\n");
    }
    return 0;
}



void read_tritfile(FILE *ptr){
    printf("File Decoding ..........\n");
    unsigned long long int tmp_8byte_decode = 0;
    size_t bufsize = NUM_TRITS+1;// Null Character
    char * buffer;
    buffer = (char*)malloc(bufsize*sizeof(char));
    if (buffer == NULL){
        printf("Error Unable to allocate buffer");
        exit(1);
    }
    int line_num = 0;
    char garbage;
    while ( fgets( buffer,bufsize, ptr) != NULL){
        garbage = fgetc(ptr);
        tmp_8byte_decode = 0;
        for(int t_idx = NUM_TRITS-1 ; t_idx >= 0  ; --t_idx ){ // encode to trits and write into trits
            tmp_8byte_decode *= 3;
            tmp_8byte_decode += buffer[t_idx]-'0';
        } 
        char chr_tmp;
        for(int shf_idx =7; shf_idx >= 0 ; --shf_idx){// take out 8 bit by 8 bit
            chr_tmp =  tmp_8byte_decode >> (shf_idx*8) & 0xff; // after shifting , taking 8 bit out
            printf("%c",chr_tmp);
        }
    }
    free(buffer);
}
