#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "bit2tritCon/char2trit_lib.h"
#include <hip/hip_runtime.h>//**************************************************
#include <hip/hip_runtime.h>//**************************************************
//#define DBG 0
int N = 7;
int p = 3;
int q =128;
int d = 2;

int g        []=  { 0,-1,-1, 0, 1, 0, 1};
int f        []=  {-1, 0, 1, 1,-1, 0, 1};
int f_inv_q  []=  {37, 2,40,21,31,26, 8};
int f_inv_p  []=  { 1, 1, 1, 1, 0, 2, 1};
int key_pub  []=  {30,26, 8,38, 2,40,20};
int rand_sel []=  {-1, 1, 0, 0, 0,-1, 1};
int irr_l    []=  {-1, 0, 0, 0, 0, 0, 0, 1};

int message  []=  {1 ,-1, 1, 1, 0,-1   };

typedef struct Poly Poly;// forward declaration

typedef void  (*func_p)(Poly *);
typedef void  (*func_p_arr_i)(Poly *,int [], int);
typedef Poly* (*func_p_i)(Poly*, int);
typedef Poly* (*func_p_p_i)(Poly *, Poly *, int);
typedef Poly* (*func_p_p_p_i)(Poly *, Poly *, Poly *,int);

struct Poly {
    int * coef;
    int degree;
    func_p free,print;
    func_p_i scalar_mult,center_lift;
    func_p_p_i add;
    func_p_p_p_i mult;
    func_p_arr_i set;
};


int table_size = 30;
int  poly_count = 0;
int  Poly_init(Poly** pt2pt);

void Poly_free(Poly* ptr);
void Poly_print(Poly* self);
Poly * Poly_CenterLift(Poly *ptr_a, int q);
Poly * Poly_scalar_mult(Poly * self, int multiplier);
Poly * Poly_add(Poly * self, Poly *ptr_b, int field_N);
Poly * Poly_mult(Poly *self, Poly *ptr_b, Poly* ptr_irr,int q);
void Poly_set(Poly *self, int [], int size); 
void File_export(FILE * fptr_out,Poly* self);


/* ref 
print("encryption ...")
r_ccov_h = p*P(rand_sel)*P(key_pub)+P(message)
cipher  = poly_ring_mult_over_q_with_irr(poly1_l=r_ccov_h.coef,poly2_l=[1],irr_l=irr_l,q=q)
print('cipher:',cipher)
*/
Poly ** poly_table = NULL;

__global__ void encrypt(Message * arr_trit_msg, int num_block, Cipher * poly_cipher_arr, Poly * poly_irr_l, Poly * poly_key_pub, Poly * poly_rand_sel, int NUM_TRITS, int p, int q){
	//int blk_idx = blockIdx.x*blockDim.x+threadIdx.x;
	int blk_idx=0;
	int TotalThread = blockDim.x*gridDim.x;
	int stripe = num_block / TotalThread;
	int head   = (blockIdx.x*blockDim.x + threadIdx.x)*stripe;
	int LoopLim = head+stripe;

	Poly* poly_message; 
    Poly* poly_scalmul ;
    Poly* poly_mult    ;
    Poly* poly_cipher  ;
	
	for(blk_idx=head ; blk_idx<LoopLim ; blk_idx++ ){
		if(poly_message -> coef != NULL){// has previous message
			poly_message -> free(poly_message); 
			poly_scalmul -> free(poly_scalmul); 
			poly_mult    -> free(poly_mult   ); 
			poly_cipher  -> free(poly_cipher ); 
		}
		poly_message -> set (poly_message  , arr_trit_msg[blk_idx].trit_poly, NUM_TRITS);
		for(int i = 0, i <= NUM_TRITS, i++){
			if(poly_message -> coef[i] < 0){
				poly_message -> coef[i] += 3;
			}
		}
		poly_scalmul = poly_rand_sel -> scalar_mult(poly_rand_sel, p) ;
		poly_mult    = poly_scalmul  -> mult(poly_scalmul, poly_key_pub, poly_irr_l, q);
		poly_cipher  = poly_mult -> add ( poly_mult, poly_message, q);
		
		Cipher_set(poly_cipher_arr[blk_idx].Cipher_poly, poly_cipher)
		poly_cipher_arr[blk_idx].degree = poly_cipher -> degree;
		//弄成array type => poly_cipher_arr[blk_idx]
		//File_export(fptr_out,poly_cipher)移到main;
	}
}

float GPU_kernel(Message * arr_trit_msg, int num_block, Cipher * poly_cipher, Poly * poly_irr_l, Poly * poly_key_pub, Poly * poly_rand_sel, int NUM_TRITS, int p, int q){
	
	Message * darr_trit_msg;
	Poly * dpoly_irr_l, * dpoly_key_pub, * dpoly_rand_sel;
	Cipher * dpoly_cipher;
	IndexSave* dInd;
	//int * poly_irr_l_coef, 
	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device******************
	hipMalloc((void**)&darr_trit_msg      ,sizeof(Message)*num_block);
	hipMalloc((void**)&dpoly_irr_l        ,sizeof(Poly)*1);
	hipMalloc((void**)&dpoly_irr_l_coef   ,sizeof(int)*(poly_irr_l->degree));
	hipMalloc((void**)&dpoly_key_pub      ,sizeof(Poly)*1);
	hipMalloc((void**)&dpoly_key_pub_coef ,sizeof(int)*(poly_key_pub->degree));
	hipMalloc((void**)&dpoly_rand_sel     ,sizeof(Poly)*1);
	hipMalloc((void**)&dpoly_rand_sel_coef,sizeof(int)*(poly_rand_sel->degree));
	cudaCalloc((void**)&dpoly_cipher       ,sizeof(Cipher)*num_block);//initialize
	// Allocate Memory Space on Device (for observation)
	//hipMalloc((void**)&dInd,sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated********************
	hipMemcpy(darr_trit_msg   , arr_trit_msg   , sizeof(Message)*num_block, hipMemcpyHostToDevice);
	hipMemcpy(dpoly_irr_l     , poly_irr_l     , sizeof(Poly)*1           , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_key_pub   , poly_key_pub   , sizeof(Poly)*1           , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_rand_sel  , poly_rand_sel  , sizeof(Poly)*1           , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_irr_l_coef    , poly_irr_l->coef   , sizeof(int)*(poly_irr_l->degree)   , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_key_pub_coef  , poly_key_pub->coef , sizeof(int)*(poly_key_pub->degree) , hipMemcpyHostToDevice);
	hipMemcpy(dpoly_rand_sel_coef , poly_rand_sel->coef, sizeof(int)*(poly_rand_sel->degree), hipMemcpyHostToDevice);
	dpoly_irr_l   ->coef = dpoly_irr_l_coef;
	dpoly_key_pub ->coef = dpoly_key_pub_coef;
	dpoly_rand_sel->coef = dpoly_rand_sel_coef;
	

	// Copy Data (indsave array) to device
	//hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	
	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid (2);
	dim3 dimBlock(4);
	encrypt<<<dimGrid,dimBlock>>>(darr_trit_msg, num_block, dpoly_cipher, dpoly_irr_l, dpoly_key_pub, dpoly_rand_sel, NUM_TRITS, p, q);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back******************
	hipMemcpy(poly_cipher, dpoly_cipher, sizeof(Cipher)*num_block, hipMemcpyDeviceToHost);
	
	// Release Memory Space on Device
	hipFree(darr_trit_msg);
	hipFree(dpoly_irr_l);
	hipFree(dpoly_irr_l_coef);
	hipFree(dpoly_key_pub);
	hipFree(dpoly_key_pub_coef);
	hipFree(dpoly_rand_sel);
	hipFree(dpoly_rand_sel_coef);
	hipFree(dpoly_cipher);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}

int main(int argc , char** argv){

    char * infilepath  =  argv[1];
    char * outfilepath =  argv[2];

    Message * arr_trit_msg = NULL;
    int num_block=0;
    FILE * fptr_out = NULL;
    fptr_out = fopen(outfilepath,"w");
	
    if( (num_block = char2trit(infilepath, arr_trit_msg)) == -1 ){
        printf("Error\n");
        return 1;
    }

    poly_table = malloc(sizeof(Poly*) * table_size);
    // param shared
    Poly * poly_irr_l  ; 
    Poly_init(& poly_irr_l   );
    poly_irr_l      -> set (poly_irr_l    ,irr_l    ,sizeof(irr_l    )/sizeof(int));
    // param decryption
    Poly * poly_g;
    Poly * poly_f      ; 
    Poly * poly_f_inv_q; 
    Poly * poly_f_inv_p; 

    Poly_init(& poly_g       );
    Poly_init(& poly_f       );
    Poly_init(& poly_f_inv_q );
    Poly_init(& poly_f_inv_p );

    poly_g          -> set (poly_g        ,g        ,sizeof(g        )/sizeof(int));
    poly_f          -> set (poly_f        ,f        ,sizeof(f        )/sizeof(int));
    poly_f_inv_q    -> set (poly_f_inv_q  ,f_inv_q  ,sizeof(f_inv_q  )/sizeof(int));
    poly_f_inv_p    -> set (poly_f_inv_p  ,f_inv_p  ,sizeof(f_inv_p  )/sizeof(int));
    // param eyncryption
    Poly * poly_key_pub; 
    //Poly * poly_message; 
    Poly * poly_rand_sel;

    Poly_init(& poly_key_pub );
    Poly_init(& poly_message );
    Poly_init(& poly_rand_sel);

    poly_key_pub    -> set (poly_key_pub  ,key_pub  ,sizeof(key_pub  )/sizeof(int));
//    poly_message    -> set (poly_message  ,message  ,sizeof(message  )/sizeof(int));
    poly_rand_sel   -> set (poly_rand_sel ,rand_sel ,sizeof(rand_sel )/sizeof(int));

	/* CPU side*/
	function_1(C,A);//to modified
	
	Cipher * arr_cipher_cpu;
	Malloc((void**)&arr_cipher_cpu,sizeof(Cipher)*num_block);
	
	/* GPU side*/
	float elapsedTime = GPU_kernel(arr_trit_msg, num_block, arr_cipher_cpu, poly_irr_l, poly_key_pub, poly_rand_sel, NUM_TRITS, p, q);
	
	printf("GPU time = %5.2f ms\n", elapsedTime);
	
	for( int blk_idx =0; blk_idx < num_block ; ++blk_idx  ){
	File_export(fptr_out,arr_cipher_cpu[blk_idx]);
	}
	
	free(arr_cipher_cpu);
//  Encryption intermediate ptr

    /*Poly* poly_scalmul ;
    Poly* poly_mult    ;
    Poly* poly_cipher  ;

    for( int blk_idx =0; blk_idx < num_block ; ++blk_idx  ){
        if(poly_message -> coef != NULL){// has previous message
             poly_message -> free(poly_message); 
             poly_scalmul -> free(poly_scalmul); 
             poly_mult    -> free(poly_mult   ); 
             poly_cipher  -> free(poly_cipher ); 
        }
        poly_message -> set (poly_message  , arr_trit_msg[blk_idx].trit_poly, NUM_TRITS);
        poly_scalmul = poly_rand_sel -> scalar_mult(poly_rand_sel, p) ;
        poly_mult    = poly_scalmul  -> mult(poly_scalmul, poly_key_pub, poly_irr_l, q);
        poly_cipher  = poly_mult -> add ( poly_mult, poly_message, q);
        File_export(fptr_out,poly_cipher);
    }*/
//    printf("message:  ");
//    poly_message -> print(poly_message);
//    Poly* poly_scalmul = poly_rand_sel -> scalar_mult(poly_rand_sel, p) ;
//    Poly* poly_mult    = poly_scalmul  -> mult(poly_scalmul, poly_key_pub, poly_irr_l, q);

//    printf("mult:  ");
//    poly_mult -> print(poly_mult);

//    Poly* poly_cipher = poly_mult -> add ( poly_mult, poly_message, q);
//    printf("Cipher:  ");
//    poly_cipher -> print(poly_cipher);

/*
    printf("Decryption:\n");
    Poly* poly_fq_mult_e = poly_cipher -> mult( poly_cipher, poly_f , poly_irr_l, q );
    printf("Poly fq mult cipher\n");
    poly_fq_mult_e -> print(poly_fq_mult_e);
    printf("Center Lifting\n");
    Poly* poly_centerlift_q = poly_fq_mult_e -> center_lift(poly_fq_mult_e, q);
    poly_centerlift_q -> print(poly_centerlift_q);
    printf("Poly f_inv_q mult center_lift q\n");
    Poly* f_inv_p_center =  poly_centerlift_q  -> mult( poly_centerlift_q, poly_f_inv_p, poly_irr_l, p);
    f_inv_p_center -> print(f_inv_p_center);
    printf("Center Lifting\n");
    Poly* decrypted =  f_inv_p_center -> center_lift(f_inv_p_center, p);
    decrypted -> print(decrypted);
*/


    // free the polynomial
    while(poly_count){
        int idx = --poly_count;
	poly_table[idx] -> free(poly_table[idx]);
        poly_table[idx] = NULL;
    }
    fclose(fptr_out);

    return 0;
}





/* ref
typedef struct {
    int * coef;
    int degree;
    func_p free,
           print;
    func_p_i scalar_mult;
    func_p_p_i add;
    func_p_p_p_i mult;
}Poly;
*/


int Poly_init(Poly** self){
    if(NULL == (*self= malloc(sizeof(Poly))) ) return EXIT_FAILURE;

    poly_table[poly_count] = (*self);
    poly_count++;
    if(poly_count == table_size){
	    printf("Table is full\n");
        Poly ** new_table = NULL;
        new_table = malloc(sizeof(Poly*) * table_size*2);
        table_size*=2;
        for(int idx=0; idx < poly_count ; ++idx){
            new_table[idx] = poly_table[idx];
        }
        free(poly_table);
        poly_table = NULL;
        poly_table = new_table;
    }
    

    (*self) ->coef   = NULL;
    (*self) ->degree = 0;

    (*self) ->free = Poly_free;   
    (*self) ->print = Poly_print;
    (*self) ->scalar_mult = Poly_scalar_mult;
    (*self) ->add   = Poly_add;
    (*self) ->mult  = Poly_mult;
    (*self) ->set   = Poly_set;
    (*self) ->center_lift= Poly_CenterLift;
    return 0;
}




void Poly_free(Poly* ptr){
    if(ptr->coef != NULL ){
        free(ptr->coef);
        ptr->coef = NULL;
        ptr->degree = 0;
    }
}
void Poly_print(Poly* self){
    //printf("Poly Coef:   ");
    //printf("{");
    for(int idx=0; idx <= self->degree ; ++idx){
        if(idx != self->degree ){
            printf("%d ",self ->coef[idx]);
        }
        else{
            printf("%d\n",self->coef[idx]);
        }
    }
}

void File_export(FILE * fptr_out,Poly* self){
    for(int idx=0; idx <= self->degree ; ++idx){
        if(idx != self->degree ){
            fprintf(fptr_out,"%d ",self ->coef[idx]);
        }
        else{
            fprintf(fptr_out,"%d\n",self->coef[idx]);
        }
    }
}

Poly * Poly_scalar_mult(Poly * self, int multiplier){
    Poly * rtn;
    Poly_init(&rtn);   
    rtn -> coef = malloc( sizeof(int)*(self->degree+1) );
    rtn -> degree = self->degree;
    for(int idx=0;idx <= self->degree ; ++idx){
        rtn->coef[idx]= (multiplier * self->coef[idx])%N;
    }
    return rtn;
}

Poly * Poly_add(Poly * ptr_a, Poly *ptr_b, int field_N){
    Poly * large_ptr = (ptr_a -> degree > ptr_b -> degree ) ? ptr_a  : ptr_b ;
    Poly * small_ptr = (ptr_a -> degree > ptr_b -> degree ) ? ptr_b : ptr_a;

    Poly * rtn;
    Poly_init(&rtn);   
    rtn -> coef = malloc( sizeof(int)*(large_ptr->degree+1) );
    rtn -> degree = large_ptr->degree;

    for(int idx =0 ; idx <= large_ptr -> degree ; ++idx){
        if ( idx > small_ptr-> degree)
            rtn -> coef[idx] = large_ptr -> coef[idx];
        else
            rtn -> coef[idx] = (large_ptr->coef[idx]+ small_ptr->coef[idx]) % field_N;
    }
    return rtn;
}

Poly * Poly_CenterLift(Poly *ptr_a, int q){

    Poly * rtn;
    Poly_init(&rtn);   
    rtn -> coef = malloc( sizeof(int)*(ptr_a->degree + 1) );
    rtn -> degree = ptr_a -> degree;

    int tmp;
    for(int i =0 ; i <= (ptr_a -> degree) ; ++i){
        tmp = ptr_a->coef[i];
        if( tmp > (q/2))
            rtn -> coef[i] = tmp - q;
        else 
            rtn -> coef[i] = tmp ;
    }
    return rtn;
}

Poly * Poly_mult(Poly *ptr_a, Poly *ptr_b, Poly* ptr_irr,int q){

    int rtn_idx = 0;
    int N = ptr_irr -> degree;
    int tmp;


    Poly * poly_rtn;
    Poly_init(& poly_rtn);
    poly_rtn -> coef = malloc(sizeof(int)*N);
    poly_rtn -> degree = N-1;

#ifdef DBG
    printf("poly rnt:"); Poly_print(poly_rtn);
    printf("ptr_a");     Poly_print(ptr_a);
    printf("ptr_b");     Poly_print(ptr_b);
    printf("ptr_irr");   Poly_print(ptr_irr);
#endif

    for (int idx = 0 ; idx <= ptr_a -> degree ; ++idx){
        for(int idy = 0 ; idy <= ptr_b -> degree ; ++ idy){
            rtn_idx = (idx + idy)%N;
#ifdef DBG
            printf("\n(%d + %d * %d )mod N ",  poly_rtn -> coef[rtn_idx], ptr_a -> coef[idx], ptr_b->coef[idy]);
#endif
            tmp  = (poly_rtn->coef[rtn_idx] + ptr_a->coef[idx] * ptr_b->coef[idy])%q;
            if(tmp <0)
                poly_rtn->coef[rtn_idx] = tmp + q;
            else
                poly_rtn->coef[rtn_idx] = tmp;
#ifdef DBG
            printf("poly_rtn[%d] = %d \n", rtn_idx, poly_rtn -> coef[rtn_idx]);
#endif
        }
    }
    // update degree
    for (int idx = N-1 ; idx >= 0 ; idx--){
	if(poly_rtn->coef[idx] != 0 ){
	    poly_rtn->degree = idx;
	    break;
        }
    }
    return poly_rtn;
}
void Poly_set(Poly *self, int arr[], int size){
    for (int idx=size-1 ; idx >=0 ; -- idx){
        if(arr[idx] != 0){
            self -> degree = idx;
            break;
        }
    }
    self->coef = malloc(sizeof(int)*size );
    for (int i = 0 ; i <= self->degree ; ++i){
        self->coef[i]=arr[i];
    }
} 

void Cipher_set(int * arr, Poly *self){
    for (int i = 0 ; i <= self->degree ; ++i){
        arr[i]=self->coef[i];
    }
} 

typedef struct Cipher {
    int Cipher_poly[251] = {0} ;
    int degree;
} Cipher;
